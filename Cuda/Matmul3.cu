#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // For hipMemcpy and hipFree
#include <torch/torch.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/autocast_mode.h>
// #include <torch/extension.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <chrono>

#include <hip/hip_fp16.h> // Include CUDA half-precision definitions



// General AtomicAdd_
template<typename T>
__device__ void AtomicAdd_(T* address, T val) {
    atomicAdd(address, val);
}
// Specialization for half precision
template<>
__device__ void AtomicAdd_(at::Half* address, at::Half val) {
    atomicAdd(reinterpret_cast<__half*>(address), *reinterpret_cast<__half*>(&val));
}
// Specialization for bfloat16 half precision
template<>
__device__ void AtomicAdd_(at::BFloat16* address, at::BFloat16 val) {
    atomicAdd(reinterpret_cast<__hip_bfloat16*>(address), *reinterpret_cast<__hip_bfloat16*>(&val));
}




// General __shfl_down_sync
template<typename T>
__device__ T __shfl_down_sync_(unsigned mask, T val, int delta, int width = warpSize) {
    return __shfl_down_sync(mask, val, delta, width);
}
// Specialization for half precision
template<>
__device__ at::Half __shfl_down_sync_(unsigned mask, at::Half val, int delta, int width) {
    return __shfl_down_sync(mask, *reinterpret_cast<__half*>(&val), delta, width);
}
// Specialization for bfloat16 half precision
template<>
__device__ at::BFloat16 __shfl_down_sync_(unsigned mask, at::BFloat16 val, int delta, int width) {
    return __shfl_down_sync(mask, *reinterpret_cast<__hip_bfloat16*>(&val), delta, width);
}





template<typename T>
__global__ void forward_kernel(
    const T* A, const T* B,
    T* output,
    int N, int M, int d
    ) {

    int block_size_x = 32;
    int block_size_y = 32;
    
    // Block indices
    int block_idx_x = blockIdx.x * block_size_x;
    int block_idx_y = blockIdx.y * block_size_y;

    // Thread indices
    int thread_idx_x = threadIdx.x; // thread in the block on the x-axis
    int thread_idx_y = threadIdx.y; // thread in the block on the y-axis


    // Calculate the block-thread index
    int block_thread_idx_x = block_idx_x + thread_idx_x;
    int block_thread_idx_y = block_idx_y + thread_idx_y;


    // Shared memory for reduction
    extern __shared__ __align__(sizeof(T)) unsigned char shared_memory_uchar[];T *shared_mem = reinterpret_cast<T *>(shared_memory_uchar);

    // Sum of the product of the elements
    T sum = 0;

    // Iterate over the dimension
    for (int d_idx = 0; d_idx < d; d_idx++) {
        // Multiply the elements, add to the sum of this output element
        sum += A[block_thread_idx_x * d + d_idx] * B[block_thread_idx_y * d + d_idx];
    }

    // Store the sum in the output matrix at output[A_idx, B_idx]
    output[block_thread_idx_x * M + block_thread_idx_y] = sum;
}



// Wrapper function to orchestrate the computation
template<typename T>
void forward(
    const T* A, const T* B, T* output,
    int N, int M, int d,
    hipStream_t stream = 0) {
    
    dim3 grid(N/32, M/32); // Number of "blocks" is how many groups of 32x32 threads we want.
    dim3 block(32, 32); // Numebr of threads per "block" will be 32x32 - the max number of threads.
    int shared_memory_size = 0; // No shared memory being used. We will add this later :)
    
    // Luanch the kernel and do the computation
    forward_kernel<T><<<grid, block, shared_memory_size, stream>>>(A, B, output, N, M, d);
}




// C++ interface
template<typename dtype_>
torch::Tensor DotProduct(torch::Tensor& A, torch::Tensor& B) {
    // Must be a CUDA tensor
    TORCH_CHECK(A.device().is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.device().is_cuda(), "B must be a CUDA tensor");

    // Get tensor dimension
    int N = A.size(0);
    int M = B.size(0);
    int d = A.size(1);

    // B should be the same size as A
    TORCH_CHECK(B.size(1) == d, "B must have the same dimension size as A");

    // Get the data type, could be auto casted
    auto data_type = at::autocast::is_enabled() && A.scalar_type() == at::kFloat ? at::kHalf : A.scalar_type();

    // Ensure the tensors are contiguous
    A = A.contiguous().to(data_type);
    B = B.contiguous().to(data_type);

    // Create the output tensor
    torch::Tensor output = torch::zeros({N, M}, torch::TensorOptions().dtype(data_type).device(A.device()));

    // https://github.com/Dao-AILab/flash-attention/blob/main/csrc/flash_attn/flash_api.cpp
    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)A.get_device()};

    // Call the CUDA kernel. The kernel is templated to handle different data types such as half, float, double.
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(A.scalar_type(), "forward_cuda", ([&] {
        forward<scalar_t>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            N, M, d);
    }));

    // Device syc for debugging
    hipDeviceSynchronize();

    return output;
}


TORCH_LIBRARY_IMPL(TORCH_EXTENSION_NAME, Autocast, m) {
    m.impl("float32", DotProduct<float>);
    m.impl("float64", DotProduct<double>);
    m.impl("float16", DotProduct<at::Half>);
    try {
        m.impl("bfloat16", DotProduct<at::BFloat16>);
    } catch (const std::exception& e) {
        std::cout << "GPU does not support bfloat16. Skipping..." << std::endl;
    }
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("float32", &DotProduct<float>);
    m.def("float64", &DotProduct<double>);
    m.def("float16", &DotProduct<at::Half>);
    try {
        m.def("bfloat16", &DotProduct<at::BFloat16>);
    } catch (const std::exception& e) {
        std::cout << "GPU does not support bfloat16. Skipping..." << std::endl;
    }
}