#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // For hipMemcpy and hipFree
#include <torch/torch.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/autocast_mode.h>
// #include <torch/extension.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <chrono>

#include <hip/hip_fp16.h> // Include CUDA half-precision definitions



// General AtomicAdd_
template<typename T>
__device__ void AtomicAdd_(T* address, T val) {
    atomicAdd(address, val);
}
// Specialization for half precision
template<>
__device__ void AtomicAdd_(at::Half* address, at::Half val) {
    atomicAdd(reinterpret_cast<__half*>(address), *reinterpret_cast<__half*>(&val));
}
// Specialization for bfloat16 half precision
template<>
__device__ void AtomicAdd_(at::BFloat16* address, at::BFloat16 val) {
    atomicAdd(reinterpret_cast<__hip_bfloat16*>(address), *reinterpret_cast<__hip_bfloat16*>(&val));
}




// General __shfl_down_sync
template<typename T>
__device__ T __shfl_down_sync_(unsigned mask, T val, int delta, int width = warpSize) {
    return __shfl_down_sync(mask, val, delta, width);
}
// Specialization for half precision
template<>
__device__ at::Half __shfl_down_sync_(unsigned mask, at::Half val, int delta, int width) {
    return __shfl_down_sync(mask, *reinterpret_cast<__half*>(&val), delta, width);
}
// Specialization for bfloat16 half precision
template<>
__device__ at::BFloat16 __shfl_down_sync_(unsigned mask, at::BFloat16 val, int delta, int width) {
    return __shfl_down_sync(mask, *reinterpret_cast<__hip_bfloat16*>(&val), delta, width);
}





template<typename T>
__global__ void forward_kernel(
    const T* A, const T* B,
    T* output,
    int N
    ) {
    
    // Not used in this case
    int blk = blockIdx.x; // Block index
    int thr = threadIdx.x; // Thread index

    // Iterate over all the elements, sum them up, and store the result in the output tensor
    for (int i = 0; i < N; i++) {
        output[0] += A[i] * B[i];
    }
}



// Wrapper function to orchestrate the computation
template<typename T>
void forward(
    const T* A, const T* B, T* output,
    int N,
    hipStream_t stream = 0) {
    
    dim3 grid(1); // Number of "blocks"
    dim3 block(1); // Numebr of threads per "block" or "blocksize"

    int shared_memory_size = 1; // Size of shared memory in bytes
    
    // Luanch the kernel and do the computation
    forward_kernel<T><<<grid, block, shared_memory_size, stream>>>(A, B, output, N);
}




// C++ interface
template<typename dtype_>
torch::Tensor DotProduct(torch::Tensor& A, torch::Tensor& B) {
    // Must be a CUDA tensor
    TORCH_CHECK(A.device().is_cuda(), "A must be a CUDA tensor");
    TORCH_CHECK(B.device().is_cuda(), "B must be a CUDA tensor");

    // Get tensor dimension
    int N = A.size(0);

    // Get the data type, could be auto casted
    auto data_type = at::autocast::is_enabled() && A.scalar_type() == at::kFloat ? at::kHalf : A.scalar_type();

    // Ensure the tensors are contiguous
    A = A.contiguous().to(data_type);
    B = B.contiguous().to(data_type);

    // Create the output tensor
    torch::Tensor output = torch::zeros({1}, torch::TensorOptions().dtype(data_type).device(A.device()));

    // https://github.com/Dao-AILab/flash-attention/blob/main/csrc/flash_attn/flash_api.cpp
    // Otherwise the kernel will be launched from cuda:0 device
    // Cast to char to avoid compiler warning about narrowing
    at::cuda::CUDAGuard device_guard{(char)A.get_device()};

    // Call the CUDA kernel. The kernel is templated to handle different data types such as half, float, double.
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(A.scalar_type(), "forward_cuda", ([&] {
        forward<scalar_t>(
            A.data_ptr<scalar_t>(),
            B.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            N);
    }));

    // Device syc for debugging
    hipDeviceSynchronize();

    return output;
}


TORCH_LIBRARY_IMPL(TORCH_EXTENSION_NAME, Autocast, m) {
    m.impl("float32", DotProduct<float>);
    m.impl("float64", DotProduct<double>);
    m.impl("float16", DotProduct<at::Half>);
    try {
        m.impl("bfloat16", DotProduct<at::BFloat16>);
    } catch (const std::exception& e) {
        std::cout << "GPU does not support bfloat16. Skipping..." << std::endl;
    }
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("float32", &DotProduct<float>);
    m.def("float64", &DotProduct<double>);
    m.def("float16", &DotProduct<at::Half>);
    try {
        m.def("bfloat16", &DotProduct<at::BFloat16>);
    } catch (const std::exception& e) {
        std::cout << "GPU does not support bfloat16. Skipping..." << std::endl;
    }
}